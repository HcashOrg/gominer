#include "hip/hip_runtime.h"
/**
 * Blake-256 HcashOrg 180-Bytes input Cuda Kernel (Tested on SM 5/5.2/6.1)
 *
 * Tanguy Pruvot - Feb 2016
 *
 * Merged 8-round blake (XVC) tweaks
 * Further improved by: ~2.72%
 * Alexis Provos - Jun 2016
 */

// nvcc  -I. -c HcashOrg.cu --ptx

#include <stdint.h>
#include <memory.h>
#include "miner.h"

#if defined(_WIN32)
#define DLLEXPORT __declspec(dllexport)
#else
#define DLLEXPORT
#endif /* _WIN32 */

extern "C" {
#include "sph/sph_blake.h"
}

/* threads per block */
#define TPB 640

/* max count of found nonces in one call (like sgminer) */
#define maxResults 4

/* hash by cpu with blake 256 */
extern "C" void HcashOrg_hash(void *output, const void *input)
{
    printf("extern \"C\" void HcashOrg_hash(void *output, const void *input)  1111111111111\n");
	sph_blake256_context ctx;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 180);
	sph_blake256_close(&ctx, output);
}

#include "cuda_helper.h"

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#define atomicInc(p, max) (*p)++
#endif

__constant__ uint32_t _ALIGN(16) c_h[2];
__constant__ uint32_t _ALIGN(16) c_data[32];
__constant__ uint32_t _ALIGN(16) c_xors[215];

#define ROR8(a)  __byte_perm(a, 0, 0x0321)
#define ROL16(a) __byte_perm(a, 0, 0x1032)

/* macro bodies */
#define pxorGS(a,b,c,d) { \
	v[a]+= c_xors[i++] + v[b]; \
	v[d] = ROL16(v[d] ^ v[a]); \
	v[c]+= v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 12); \
	v[a]+= c_xors[i++] + v[b]; \
	v[d] = ROR8(v[d] ^ v[a]); \
	v[c]+= v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
}

#define pxorGS2(a,b,c,d, a1,b1,c1,d1) {\
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROL16(v[ d] ^ v[ a]);           v[d1] = ROL16(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12);      v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROR8(v[ d] ^ v[ a]);            v[d1] = ROR8(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7);       v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

#define pxory1GS2(a,b,c,d, a1,b1,c1,d1) { \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROL16(v[ d] ^ v[ a]);           v[d1] = ROL16(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12);      v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= (c_xors[i++]^nonce) + v[b1]; \
	v[ d] = ROR8(v[ d] ^ v[ a]);            v[d1] = ROR8(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7);       v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

#define pxory0GS2(a,b,c,d, a1,b1,c1,d1) { \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROL16(v[ d] ^ v[ a]);           v[d1] = ROL16(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12);      v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= (c_xors[i++]^nonce) + v[ b];    v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROR8(v[ d] ^ v[ a]);            v[d1] = ROR8(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7);       v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

#define pxorx1GS2(a,b,c,d, a1,b1,c1,d1) { \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= (c_xors[i++]^nonce) + v[b1]; \
	v[ d] = ROL16(v[ d] ^ v[ a]);           v[d1] = ROL16(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12);      v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROR8(v[ d] ^ v[ a]);            v[d1] = ROR8(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7);       v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

#define pxorx0GS2(a,b,c,d, a1,b1,c1,d1) { \
	v[ a]+= (c_xors[i++]^nonce) + v[ b];    v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROL16(v[ d] ^ v[ a]); 	        v[d1] = ROL16(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12);      v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= c_xors[i++] + v[ b]; 			v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROR8(v[ d] ^ v[ a]); 	        v[d1] = ROR8(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7); 		v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

extern "C"
{

//__global__ __launch_bounds__(TPB,1)
__global__ void HcashOrg_gpu_hash_nonce(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint32_t highTarget)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	if (thread < threads)
	{
		uint32_t v[16];
		#pragma unroll
		for(int i=0; i<16; i+=4) {
			*(uint4*)&v[i] = *(uint4*)&c_data[i];
		}

		const uint32_t nonce = startNonce + thread;
		v[ 1]+= (nonce ^ 0x13198A2E);
		v[13] = ROR8(v[13] ^ v[1]);
		v[ 9]+= v[13];
		v[ 5] = ROTR32(v[5] ^ v[9], 7);

		int i = 0;
		v[ 1]+= c_xors[i++];// + v[ 6];
		v[ 0]+= v[5];
		v[12] = ROL16(v[12] ^ v[ 1]);         v[15] = ROL16(v[15] ^ v[ 0]);
		v[11]+= v[12];                        v[10]+= v[15];
		v[ 6] = ROTR32(v[ 6] ^ v[11], 12);    v[ 5] = ROTR32(v[5] ^ v[10], 12);
		v[ 1]+= c_xors[i++] + v[ 6];          v[ 0]+= c_xors[i++] + v[ 5];
		v[12] = ROR8(v[12] ^ v[ 1]);          v[15] = ROR8(v[15] ^ v[ 0]);
		v[11]+= v[12];                        v[10]+= v[15];
		v[ 6] = ROTR32(v[ 6] ^ v[11], 7);     v[ 5] = ROTR32(v[ 5] ^ v[10], 7);

		pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxory1GS2( 2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorx1GS2( 0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorx1GS2( 0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorx1GS2( 2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxory1GS2( 2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxory1GS2( 0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorx1GS2( 2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxory0GS2( 2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorx0GS2( 2, 7, 8, 13, 3, 4, 9, 14);
		pxory1GS2( 0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxory1GS2( 2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorx1GS2( 0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorx1GS2( 0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorGS(    2, 7, 8, 13);

		if ((c_h[1]^v[15]) == v[7]) {
		        uint32_t pos = atomicInc(&resNonce[0], UINT32_MAX)+1;
			resNonce[pos] = nonce;
			return;
		}
	}
}
}

extern "C" {
DLLEXPORT void
HcashOrg_hash_nonce(uint32_t grid, uint32_t block, uint32_t threads,
    uint32_t startNonce, uint32_t *resNonce, uint32_t targetHigh)
{
	HcashOrg_gpu_hash_nonce <<<grid, block>>> (threads, startNonce, resNonce, targetHigh);
}
}

extern "C" {
__host__ DLLEXPORT void
HcashOrg_cpu_setBlock_52(const uint32_t *input, uint32_t updateHeight)
{
/*
    printf("HcashOrg_cpu_setBlock_52  1111111:\n");
	for (int i = 0; i < 244/4; i++)
		printf("%08x", input[i]);
	printf("  \nHcashOrg_cpu_setBlock_52 end 2222222222\n");

    fflush(stdout);

	Precompute everything possible and pass it on constant memory
*/
	const uint32_t z[16] = {
		0x243F6A88U, 0x85A308D3U, 0x13198A2EU, 0x03707344U,
		0xA4093822U, 0x299F31D0U, 0x082EFA98U, 0xEC4E6C89U,
		0x452821E6U, 0x38D01377U, 0xBE5466CFU, 0x34E90C6CU,
		0xC0AC29B7U, 0xC97C50DDU, 0x3F84D5B5U, 0xB5470917U
	};

    uint32_t height = input[128/4];
 //   printf("height = %d, updateHeight = %d\n", height, updateHeight);
	int i=0;
	uint32_t _ALIGN(64) preXOR[215];
	uint32_t _ALIGN(64)   data[16];
	uint32_t _ALIGN(64)      m[16];
	uint32_t _ALIGN(64)      h[ 2];

	sph_blake256_context ctx;
	sph_blake256_set_rounds(14);
	sph_blake256_init(&ctx);
	if(height < updateHeight){
	    sph_blake256(&ctx, input, 128);
	}else{
	    sph_blake256(&ctx, input, 192);
	}


	data[ 0] = ctx.H[0];
	data[ 1] = ctx.H[1];
	data[ 2] = ctx.H[2];
	data[ 3] = ctx.H[3];
	data[ 4] = ctx.H[4];
	data[ 5] = ctx.H[5];
	data[ 8] = ctx.H[6];

#define  BLOCK_OFFSET 16
    if(height < updateHeight){
	    data[12] = swab32(input[35]);
	}else{
	    data[12] = swab32(input[35 + BLOCK_OFFSET]);
	}
	data[13] = ctx.H[7];


	// pre swab32
	if(height < updateHeight){
        m[ 0] = swab32(input[32]);	m[ 1] = swab32(input[33]);
        m[ 2] = swab32(input[34]);	m[ 3] = 0;
        m[ 4] = swab32(input[36]);	m[ 5] = swab32(input[37]);
        m[ 6] = swab32(input[38]);	m[ 7] = swab32(input[39]);
        m[ 8] = swab32(input[40]);	m[ 9] = swab32(input[41]);
        m[10] = swab32(input[42]);	m[11] = swab32(input[43]);
        m[12] = swab32(input[44]);	m[13] = 0x80000001;
	}else{
	    m[ 0] = swab32(input[32 + BLOCK_OFFSET]);	m[ 1] = swab32(input[33 + BLOCK_OFFSET]);
    	m[ 2] = swab32(input[34 + BLOCK_OFFSET]);	m[ 3] = 0;
    	m[ 4] = swab32(input[36 + BLOCK_OFFSET]);	m[ 5] = swab32(input[37 + BLOCK_OFFSET]);
    	m[ 6] = swab32(input[38 + BLOCK_OFFSET]);	m[ 7] = swab32(input[39 + BLOCK_OFFSET]);
    	m[ 8] = swab32(input[40 + BLOCK_OFFSET]);	m[ 9] = swab32(input[41 + BLOCK_OFFSET]);
    	m[10] = swab32(input[42 + BLOCK_OFFSET]);	m[11] = swab32(input[43 + BLOCK_OFFSET]);
    	m[12] = swab32(input[44 + BLOCK_OFFSET]);	m[13] = 0x80000001;
	}
	m[14] = 0;
	if(height < updateHeight){
		m[15] = 0x000005a0;
	}else{
		m[15] = 0x000007a0;
	}
/*
    printf("hash 123456 :\n");
    for (int i = 0; i < 8; i++)
		printf("%08x", ctx.H[i]);
	printf("\nhash 123456 :\n");
    printf("HcashOrg_cpu_setBlock_52  1111111:\n");
	for (int i = 0; i < 16; i++)
		printf("%08x", m[i]);
	printf("  \nHcashOrg_cpu_setBlock_52 end 2222222222\n");
    fflush(stdout);
*/
	h[ 0] = data[ 8];
	h[ 1] = data[13];

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_h),h, 8, 0, hipMemcpyHostToDevice));

	data[ 0]+= (m[ 0] ^ z[1]) + data[ 4];
	data[12]  = SPH_ROTR32(z[4] ^ SPH_C32( height < updateHeight ? 0x5A0: 0x7A0) ^ data[ 0], 16);

	data[ 8] = z[0]+data[12];
	data[ 4] = SPH_ROTR32(data[ 4] ^ data[ 8], 12);
	data[ 0]+= (m[ 1] ^ z[0]) + data[ 4];
	data[12] = SPH_ROTR32(data[12] ^ data[ 0],8);
	data[ 8]+= data[12];
	data[ 4] = SPH_ROTR32(data[ 4] ^ data[ 8], 7);

	data[ 1]+= (m[ 2] ^ z[3]) + data[ 5];
	data[13] = SPH_ROTR32((z[5] ^ SPH_C32(height < updateHeight ? 0x5A0: 0x7A0)) ^ data[ 1], 16);
	data[ 9] = z[1]+data[13];
	data[ 5] = SPH_ROTR32(data[ 5] ^ data[ 9], 12);
	data[ 1]+= data[ 5]; //+nonce ^ ...

	data[ 2]+= (m[ 4] ^ z[5]) + h[ 0];
	data[14] = SPH_ROTR32(z[6] ^ data[ 2],16);
	data[10] = z[2] + data[14];
	data[ 6] = SPH_ROTR32(h[ 0] ^ data[10], 12);
	data[ 2]+= (m[ 5] ^ z[4]) + data[ 6];
	data[14] = SPH_ROTR32(data[14] ^ data[ 2], 8);
	data[10]+= data[14];
	data[ 6] = SPH_ROTR32(data[ 6] ^ data[10], 7);

	data[ 3]+= (m[ 6] ^ z[7]) + h[ 1];
	data[15] = SPH_ROTR32(z[7] ^ data[ 3],16);
	data[11] = z[3] + data[15];
	data[ 7] = SPH_ROTR32(h[ 1] ^ data[11], 12);
	data[ 3]+= (m[ 7] ^ z[6]) + data[ 7];
	data[15] = SPH_ROTR32(data[15] ^ data[ 3],8);
	data[11]+= data[15];
	data[ 7] = SPH_ROTR32(data[11] ^ data[ 7], 7);
	data[ 0]+= m[ 8] ^ z[9];

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_data), data, 64, 0, hipMemcpyHostToDevice));

#define precalcXORGS(x,y) { \
	preXOR[i++]= (m[x] ^ z[y]); \
	preXOR[i++]= (m[y] ^ z[x]); \
}
#define precalcXORGS2(x,y,x1,y1){\
	preXOR[i++] = (m[ x] ^ z[ y]);\
	preXOR[i++] = (m[x1] ^ z[y1]);\
	preXOR[i++] = (m[ y] ^ z[ x]);\
	preXOR[i++] = (m[y1] ^ z[x1]);\
}
	precalcXORGS(10,11);
	preXOR[ 0]+=data[ 6];
	preXOR[i++] = (m[9] ^ z[8]);
	precalcXORGS2(12,13,14,15);
	precalcXORGS2(14,10, 4, 8);
	precalcXORGS2( 9,15,13, 6);
	precalcXORGS2( 1,12, 0, 2);
	precalcXORGS2(11, 7, 5, 3);
	precalcXORGS2(11, 8,12, 0);
	precalcXORGS2( 5, 2,15,13);
	precalcXORGS2(10,14, 3, 6);
	precalcXORGS2( 7, 1, 9, 4);
	precalcXORGS2( 7, 9, 3, 1);
	precalcXORGS2(13,12,11,14);
	precalcXORGS2( 2, 6, 5,10);
	precalcXORGS2( 4, 0,15, 8);
	precalcXORGS2( 9, 0, 5, 7);
	precalcXORGS2( 2, 4,10,15);
	precalcXORGS2(14, 1,11,12);
	precalcXORGS2( 6, 8, 3,13);
	precalcXORGS2( 2,12, 6,10);
	precalcXORGS2( 0,11, 8, 3);
	precalcXORGS2( 4,13, 7, 5);
	precalcXORGS2(15,14, 1, 9);
	precalcXORGS2(12, 5, 1,15);
	precalcXORGS2(14,13, 4,10);
	precalcXORGS2( 0, 7, 6, 3);
	precalcXORGS2( 9, 2, 8,11);
	precalcXORGS2(13,11, 7,14);
	precalcXORGS2(12, 1, 3, 9);
	precalcXORGS2( 5, 0,15, 4);
	precalcXORGS2( 8, 6, 2,10);
	precalcXORGS2( 6,15,14, 9);
	precalcXORGS2(11, 3, 0, 8);
	precalcXORGS2(12, 2,13, 7);
	precalcXORGS2( 1, 4,10, 5);
	precalcXORGS2(10, 2, 8, 4);
	precalcXORGS2( 7, 6, 1, 5);
	precalcXORGS2(15,11, 9,14);
	precalcXORGS2( 3,12,13, 0);
	precalcXORGS2( 0, 1, 2, 3);
	precalcXORGS2( 4, 5, 6, 7);
	precalcXORGS2( 8, 9,10,11);
	precalcXORGS2(12,13,14,15);
	precalcXORGS2(14,10, 4, 8);
	precalcXORGS2( 9,15,13, 6);
	precalcXORGS2( 1,12, 0, 2);
	precalcXORGS2(11, 7, 5, 3);
	precalcXORGS2(11, 8,12, 0);
	precalcXORGS2( 5, 2,15,13);
	precalcXORGS2(10,14, 3, 6);
	precalcXORGS2( 7, 1, 9, 4);
	precalcXORGS2( 7, 9, 3, 1);
	precalcXORGS2(13,12,11,14);
	precalcXORGS2( 2, 6, 5,10);
	precalcXORGS( 4, 0);
	precalcXORGS(15, 8);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_xors), preXOR, 215*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}
}

/* ############################################################################################################################### */

